#include "hip/hip_runtime.h"
#include "gpu_velocity_grid.hpp"
using namespace spatial_cell;

// Copies velocity_block_list and block_data as well as necessary constants from a SpatialCell to GPU for processing.
GPU_velocity_grid::GPU_velocity_grid(SpatialCell *spacell) {
    cpu_cell = spacell;
    // Allocate memory
    unsigned int vel_block_list_size = spacell->number_of_blocks*sizeof(unsigned int);
    unsigned int block_data_size = spacell->block_data.size() * sizeof(Real);

    // Note that vel_grid (aka. the actual velocity space) has to allocated separately in init_grid
    CUDACALL(hipMalloc(&num_blocks, sizeof(unsigned int)));
    CUDACALL(hipMalloc(&velocity_block_list, vel_block_list_size));
    CUDACALL(hipMalloc(&block_data, block_data_size));
    CUDACALL(hipMalloc(&min_val, sizeof(Real)));
    CUDACALL(hipMalloc(&grid_dims, sizeof(grid_dims_t)));
    grid_dims_host = new grid_dims_t();
    grid_dims_host->sparse_size.x = SpatialCell::vx_length;
    grid_dims_host->sparse_size.y = SpatialCell::vy_length;
    grid_dims_host->sparse_size.z = SpatialCell::vz_length;
    grid_dims_host->cell_dv = SpatialCell::cell_dvx; // NOTE: Only one cell_dv is used for now as they are always the same in all dimensions.

    // Copy to gpu
    unsigned int *velocity_block_list_arr = &(spacell->velocity_block_list[0]);
    Real *block_data_arr = &(spacell->block_data[0]);
    num_blocks_host = spacell->number_of_blocks;
    CUDACALL(hipMemcpy(min_val, &(SpatialCell::velocity_block_min_value), sizeof(Real), hipMemcpyHostToDevice));
    CUDACALL(hipMemcpy(num_blocks, &(spacell->number_of_blocks), sizeof(unsigned int), hipMemcpyHostToDevice));
    CUDACALL(hipMemcpy(&grid_dims, &grid_dims_host, sizeof(grid_dims_t), hipMemcpyHostToDevice));
    CUDACALL(hipMemcpy(velocity_block_list, velocity_block_list_arr, vel_block_list_size, hipMemcpyHostToDevice));
    CUDACALL(hipMemcpy(block_data, block_data_arr, block_data_size, hipMemcpyHostToDevice));
}

// The proper destructor for GPU_velocity_grid that has to be called manually. See the destructor comments for details.
__host__ void GPU_velocity_grid::del(void) {
// Free memory
    CUDACALL(hipFree(vel_grid));
    CUDACALL(hipFree(num_blocks));
    CUDACALL(hipFree(velocity_block_list));
    CUDACALL(hipFree(block_data));
    CUDACALL(hipFree(min_val));
    CUDACALL(hipFree(grid_dims));
    delete grid_dims_host;
}

// Nothing in here because this is called whenever a copy-by-value goes out of scope. Call dell when you want to free memory related to the instance.
__host__ __device__ GPU_velocity_grid::~GPU_velocity_grid() {}

__global__ void print_cells_k(GPU_velocity_grid ggrid) {
    ind3d inds = {15,15,15};
    unsigned int ind = ggrid.get_velocity_block(inds);
    printf("%u %u %u: %e \n", inds.x, inds.y, inds.z, ggrid.get_velocity_cell(ind, 0));
    inds.x = 16; inds.y = 16; inds.z = 16;
    ind = ggrid.get_velocity_block(inds);
    printf("%u %u %u: %e \n", inds.x, inds.y, inds.z, ggrid.get_velocity_cell(ind, 0));inds.x = 17; inds.y = 17; inds.z = 17;
    ind = ggrid.get_velocity_block(inds);
    printf("%u %u %u: %e \n", inds.x, inds.y, inds.z, ggrid.get_velocity_cell(ind, 0));
}

// Same as SpatialCell::get_velocity_block_indices but revised for GPU. Constructs 3d indices from 1d index.
__device__ ind3d GPU_velocity_grid::get_velocity_block_indices(const unsigned int blockid) {
    ind3d indices;
    indices.x = blockid % this->grid_dims->sparse_size.x;
    indices.y = (blockid / this->grid_dims->sparse_size.x) % this->grid_dims->sparse_size.y;
    indices.z = blockid / (this->grid_dims->sparse_size.x * this->grid_dims->sparse_size.y);

    return indices;
}

__device__ ind3d GPU_velocity_grid::get_full_grid_block_indices(const unsigned int blockid) {
    ind3d indices;
    indices.x = blockid % this->grid_dims->size.x;
    indices.y = (blockid / this->grid_dims->size.x) % this->grid_dims->size.y;
    indices.z = blockid / (this->grid_dims->size.x * this->grid_dims->size.y);

    return indices;
}

__host__ ind3d GPU_velocity_grid::get_full_grid_block_indices_host(const unsigned int blockid) {
    ind3d indices;
    ind3d dims = this->grid_dims_host->size;
    indices.x = blockid % dims.x;
    indices.y = (blockid / dims.x) % dims.y;
    indices.z = blockid / (dims.x * dims.y);

    return indices;
}

// Host version. Requires initialized SpatialCell static variables.
__host__ ind3d GPU_velocity_grid::get_velocity_block_indices_host(const unsigned int blockid) {
    ind3d indices;
    indices.x = blockid % SpatialCell::vx_length;
    indices.y = (blockid / SpatialCell::vx_length) % SpatialCell::vy_length;
    indices.z = blockid / (SpatialCell::vx_length * SpatialCell::vy_length);

    return indices;
}

// Constructs 1d index out of 3d indices
__device__ unsigned int GPU_velocity_grid::get_velocity_block(const ind3d indices) {
    unsigned int ret = indices.x + indices.y * this->grid_dims->sparse_size.x + indices.z * this->grid_dims->sparse_size.x * this->grid_dims->sparse_size.y;
    //printf("%u %u %u: %u\n", indices.x, indices.y, indices.z, ret);
    return ret;
}


// Same as print_blocks, but prints from a kernel
__global__ void kernel_print_blocks(GPU_velocity_grid grid) {
    unsigned int tid = blockIdx.x;
    unsigned int ind;
    ind3d indices;
    ind = grid.velocity_block_list[tid];
    indices = grid.get_velocity_block_indices(ind);
    printf("%5.0u: (%4i, %4i, %4i) %7.1f\n", ind, indices.x, indices.y, indices.z, grid.block_data[tid*WID3]);
}

__device__ vel_block* GPU_velocity_grid::get_velocity_grid_block(unsigned int blockid) {
    ind3d block_indices = GPU_velocity_grid::get_velocity_block_indices(blockid);
    //printf("%u: %u %u %u\n", blockid, block_indices.x, block_indices.y, block_indices.z);
    // Check for out of bounds
    grid_dims_t dims = *this->grid_dims;
    if (block_indices.x > dims.max.x ||
        block_indices.y > dims.max.y ||
        block_indices.z > dims.max.z ||
        block_indices.x < dims.min.x ||
        block_indices.y < dims.min.y ||
        block_indices.z < dims.min.z) return ERROR_BLOCK;
    // Move the indices to same origin and dimensions as the bounding box
    ind3d n_ind = {block_indices.x - dims.min.x, block_indices.y - dims.min.y, block_indices.z - dims.min.z};
    vel_block *block_ptr = &vel_grid[n_ind.x + n_ind.y*dims.size.x + n_ind.z*dims.size.x*dims.size.y];
    //printf("%4u: %2u %2u %2u, %2u %2u %2u. %016lx\n", n_ind.x + n_ind.y*box_dims.x + n_ind.z*box_dims.x*box_dims.y, block_indices.x, block_indices.y, block_indices.z, n_ind.x, n_ind.y, n_ind.z, block_ptr);
    return block_ptr;
}

// Returns index of the sparse grid corresponding to the blockid of the full grid
__device__ int GPU_velocity_grid::full_to_sparse_ind(unsigned int blockid) {
    ind3d full_inds = get_full_grid_block_indices(blockid);
    ind3d sparse_inds = {this->grid_dims->min.x + full_inds.x, this->grid_dims->min.y + full_inds.y, this->grid_dims->min.z + full_inds.z};
    return sparse_inds.x + sparse_inds.y * this->grid_dims->sparse_size.x + sparse_inds.z * this->grid_dims->sparse_size.x * this->grid_dims->sparse_size.y;
}

// Same as above for host. Requires indices of the minimum point of the full grid.
__host__ int GPU_velocity_grid::full_to_sparse_ind_host(unsigned int blockid) {
        ind3d mins = this->grid_dims_host->min;
        ind3d full_inds = this->get_full_grid_block_indices_host(blockid);
        ind3d sparse_inds = {mins.x + full_inds.x, mins.y + full_inds.y, mins.z + full_inds.z};
    return sparse_inds.x + sparse_inds.y * SpatialCell::vx_length + sparse_inds.z * SpatialCell::vx_length * SpatialCell::vy_length;
}

// Returns the data from a given block and cell id.
__device__ Real GPU_velocity_grid::get_velocity_cell(unsigned int blockid, unsigned int cellid) {
    vel_block *block = get_velocity_grid_block(blockid);
    // Check for out of bounds
    if (block == ERROR_BLOCK) return ERROR_CELL;
    if (cellid >= WID3) return ERROR_CELL;
    //unsigned int indx = (*num_blocks)-5;
    //printf("%08lx ", &(vel_grid[0]));
    //printf("%08lx\n", &(vel_grid[indx]));
    //printf("%u %u %08lx\n", blockid, cellid, block->data);
    Real ret = block->data[cellid];
    return ret;
}

// Sets the data in a given block and cell id to val. Returns the old value of the cell.
__device__ Real GPU_velocity_grid::set_velocity_cell(unsigned int blockid, unsigned int cellid, Real val) {
    vel_block *block = get_velocity_grid_block(blockid);
    // Check for out of bounds
    if (block == ERROR_BLOCK) return ERROR_CELL;
    Real old = block->data[cellid];
    block->data[cellid] = val;
    return old;
}

// Sets the data in a given block to that of vals.
__device__ void GPU_velocity_grid::set_velocity_block(unsigned int blockid, Real *vals) {
    vel_block *block = get_velocity_grid_block(blockid);
    // Check for out of bounds
    if (block == ERROR_BLOCK) {
        printf("Error bad index in set_velocity_block: %u\n", blockid);
        return;
    }
    for (int i = 0; i < WID3; i++){
        block->data[i] = vals[i];
    }
    __syncthreads();
    return;
}

// Fills the given array of size len with val
__global__ void init_data(vel_block *grid, Real val, int len) {
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i < len) {
        for (int j = 0; j < WID3; j++) {
            grid[i].data[j] = val;
        }
    }
}

// Copies data from block_data to vel_grid
__global__ void copy_block_data(GPU_velocity_grid ggrid) {
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i < *(ggrid.num_blocks)) {
        int blockid = ggrid.velocity_block_list[i];
        ggrid.set_velocity_block(blockid, &(ggrid.block_data[i*WID3]));
    }
}

// Allocates a full velocity grid and copies data from block_data.
__host__ void GPU_velocity_grid::init_grid(void) {
    unsigned int min = this->min_ind();
    unsigned int max = this->max_ind();
    ind3d min_i = get_velocity_block_indices_host(min);
    ind3d max_i = get_velocity_block_indices_host(max);
    printf("MIN: %u %u %u %u\n", min, min_i.x, min_i.y, min_i.z);
    printf("MAX: %u %u %u %u\n", max, max_i.x, max_i.y, max_i.z);
    // dimensions of the grid
    unsigned int dx = max_i.x - min_i.x + 1;
    unsigned int dy = max_i.y - min_i.y + 1;
    unsigned int dz = max_i.z - min_i.z + 1;
    unsigned int vel_grid_len = dx*dy*dz;
    printf("GRID DIMS: %u %u %u: %u\n", dx, dy, dz, vel_grid_len);
    ind3d dims = {dx, dy, dz};

    CUDACALL(hipMalloc(&vel_grid, vel_grid_len * sizeof(vel_block)));

    // Copy constants to device
    CUDACALL(hipMemcpy(&this->grid_dims->min, &min_i, sizeof(ind3d), hipMemcpyHostToDevice));
    CUDACALL(hipMemcpy(&this->grid_dims->max, &max_i, sizeof(ind3d), hipMemcpyHostToDevice));
    CUDACALL(hipMemcpy(&this->grid_dims->size, &dims, sizeof(ind3d), hipMemcpyHostToDevice));
    
    this->grid_dims_host->min = min_i;
    this->grid_dims_host->max = max_i;
    this->grid_dims_host->size = dims;

    // Calculate grid dimensions and start kernel
    unsigned int blockSize = 64;
    unsigned int gridSize = ceilDivide(vel_grid_len, blockSize);
    init_data<<<gridSize, blockSize>>>(vel_grid, 0.0f, vel_grid_len);
    gridSize = num_blocks_host;
    printf("%u ", gridSize);
    gridSize = ceilDivide(gridSize, blockSize);
    printf("%u %u\n", gridSize, blockSize);
    CUDACALL(hipDeviceSynchronize()); // Wait for initialization to finish
    copy_block_data<<<gridSize, blockSize>>>(*this);
    CUDACALL(hipDeviceSynchronize()); // Block before returning
}

// Creates a list of booleans in allocated list "list" where list[i] is true if the block at vel_grid[i] includes a cell with a value larger than SpatialCell::velocity_block_min_value. N is the size of vel_grid.
__global__ void relevant_block_list(bool *list, int N, GPU_velocity_grid grid) {
    unsigned int tid = blockIdx.x*blockDim.x + threadIdx.x;
    if (tid < N) {
        Real min_value = *grid.min_val;
        int i;
        vel_block *block_ptr = &(grid.vel_grid[tid]);
        for (i = 0; i < WID3; i++) {
            //printf("%i %i %i %016lx %016lx\n", tid, i, N, block_ptr, block_ptr->data);
            if (block_ptr->data[i] > min_value) {
                list[tid] = true;
                break;
            }
        }
        if (i == WID3) list[tid] = false;
    }
    __syncthreads();
}

// Creates a new SpatialCell with data from the full grid on GPU
__host__ SpatialCell* GPU_velocity_grid::toSpatialCell(void) {
    SpatialCell *spacell = cpu_cell; // The input SpatialCell is used to create the output.
    ind3d bounding_box_dims, bounding_box_mins;
    bool *relevant_blocks;
    CUDACALL(hipMemcpy(&bounding_box_dims, &this->grid_dims->size, sizeof(ind3d), hipMemcpyDeviceToHost));
    CUDACALL(hipMemcpy(&bounding_box_mins, &this->grid_dims->min,  sizeof(ind3d), hipMemcpyDeviceToHost));

    int box_size = bounding_box_dims.x * bounding_box_dims.y * bounding_box_dims.z;
    CUDACALL(hipMalloc(&relevant_blocks, box_size * sizeof(bool)));
    
    const int blockSize = 64;
    const int gridSize = ceilDivide(box_size, 64);
    relevant_block_list<<<gridSize, blockSize>>>(relevant_blocks, box_size, *this);
    
    clear_data(spacell); // Remove block data but keep memory allocation. Many of the original blocks should still exist, so no need to allocate for them again.
    
    bool *rel_blocks = (bool *)malloc(box_size * sizeof(bool));
    CUDACALL(hipDeviceSynchronize());
    CUDACALL(hipMemcpy(rel_blocks, relevant_blocks, box_size * sizeof(bool), hipMemcpyDeviceToHost));
    
    unsigned int ind;
    std::vector<int> rel_block_inds;
    for (int i = 0; i < box_size; i++) {
        // See if the block should be copied.
        if (!rel_blocks[i]) continue;
        ind = this->full_to_sparse_ind_host(i);
        rel_block_inds.push_back(ind);
        // Create the block in SpatialCell
        spacell->add_velocity_block(ind);
        Velocity_Block* block_ptr = spacell->at(ind);
        // Copy the data over blockwise.
        //CUDACALL(hipMemcpyAsync(&(block_ptr->data[0]), &(vel_grid[i].data[0]), 1 * sizeof(Real), hipMemcpyDeviceToHost));
        CUDACALL(hipMemcpy(&(block_ptr->data[0]), &(vel_grid[i].data[0]), WID3 * sizeof(Real), hipMemcpyDeviceToHost));
    }
    /*
    printf("Number of relevant blocks: %4lu\n", rel_block_inds.size());
    for (int i = 0; i < rel_block_inds.size(); i++) {
        int ind = rel_block_inds[i];
        Velocity_Block* block_ptr = spacell->at(ind);
        ind3d inds = GPU_velocity_grid::get_velocity_block_indices_host(ind);
        printf(block_print_format, ind, inds.x, inds.y, inds.z, block_ptr->data[0]);
    }
    putchar('\n');
    */
    CUDACALL(hipFree(relevant_blocks));
    CUDACALL(hipDeviceSynchronize());
    return spacell;
}
